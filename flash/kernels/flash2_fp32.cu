#include "hip/hip_runtime.h"
// source: https://github.com/kilianhae/FlashAttention.C/blob/main/src/flashattention.cu
#include <torch/extension.h>
#include <torch/types.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>

#include <algorithm>
#include <cassert>
#include <cstdio>
#include <cstdlib>
#include <hipblas.h>
#include <hip/hip_runtime.h>

#define NEG_INFINITY __int_as_float(0xff800000)

# define d 64
# define B_r 32 // How many rows of Q_i are processed by one threadblock
# define B_c 32 // How many rows of K_i and V_i are processed by one threadblock
# define BK 64 // for now = B_c

// thread - 2nd level tiling
# define TM 4 // How many rows f the attention Matrix S are processed by a single thread
# define TN 4 // How many columns of the attention Matrix S are processed by a single thread

# define CACHE_Q 0

__global__
void flash_tiled_coarse(float *out, float* out_l, float *K, float *Q, float* V, float scaling, int batch_stride, int T_r, int T_c, int seq_len)
{
  int tid_x = threadIdx.x;
  int tid_y = threadIdx.y;
  int batch_offset = batch_stride * blockIdx.x;

  /*
  all are fully loaded into shared memory SMEM, I think we should adjust this as second step to only loading it in tiles of B_r x 32 
  and iterating the mults over the 32 sized tiles this way we can have a larger d, while keeping occupancy high
  */
  /*
    NOTE: all are fully loaded into shared memory SMEM, I think we should adjust this as second step to only loading it in tiles of B_r x 32 
    and iterating the mults over the 32 sized tiles this way we can have a larger d, while keeping occupancy high
    */

    // statically define in SMEM and still address it with indices
    //__shared__ float Q_i[B_r][d]; // uncomment only if you want to cache over full d (if CACHE_Q = 1)
    __shared__ float Q_i[B_r][BK]; // if you want to save SMEM loads and keep the full Q loaded then change this to [B_r][d]
    
    __shared__ float K_j[B_c][BK+1]; // reduce SMEM bank conflicts by adding 1 column as K will be loaded transposed!
    __shared__ float V_j[B_c][BK];
    
    // attention result
    __shared__ float S_i[B_r][B_c+1]; // reduce SMEM bank conflicts by adding 1 column (in the naive softmax part)
    const int num_tiles = d/BK; // how many tiles are the computation of the attention is split into

    const uint totalResultsBlocktile = B_r * B_c; // number of results to calculate per block
    const uint numThreadsBlocktile = totalResultsBlocktile / (TM * TN); // number of threads needed
    const int threadId_flat = threadIdx.y * blockDim.x + threadIdx.x; // flattened thread id  (used for coalesced loading of tiles)

    // each thread process one block at position:
    const int threadCol = threadId_flat % (B_c / TN);
    const int threadRow = threadId_flat / (B_c / TN);
        
    float l_i[TM]= {0.0};; // storing the intermediate sum of exponentials per row
    float m_i[TM]; // storing the intermediate max value of the rows
    float last_m[TM]; // storing the last max value of the rows
    float O_i[num_tiles * TN * TM] = {0.0}; // storing the intermediate results of the Outputs (each thread stores a chunk TM x TN per tile)
    
    // reset to min
    for (int ii = 0; ii < TM; ii++) {
        m_i[ii] = -INFINITY;
    }

    //WARNING: due to coalsecing I should probably add a second set of variables for using BK+1
    const uint strideK = numThreadsBlocktile / BK; // 64 / 64 = 1
    const uint innerRowK = threadId_flat / BK; // 0-63 / 64, 0000000000000...0
    const uint innerColK = threadId_flat % BK; // 0-63 % 64, 0123456789101112...63

    int id;
    // load Q_i, UNCOMMENT only if your Q is caching over full d
    const uint innerRowQ = threadId_flat / d; // 0-63 / 64, 0000000000000...0
    const uint innerColQ = threadId_flat % d; // 0-63 % 64, 0123456789012...63
    const uint nr_loads = B_r * d / numThreadsBlocktile;

    for (int t=0; t<nr_loads; t++){
      // need to load block of size B_r x d (64 x 64) with numThreadsBlocktile threads
      // if (blockIdx.y * B_r + innerRowQ) * d + innerColQ + t * numThreadsBlocktile / d
      id = (blockIdx.y * B_r + innerRowQ) * d + innerColQ + t * numThreadsBlocktile;
      // 4 x 4 then this is 5 thus 5/
      if (id < d*seq_len){
        Q_i[innerRowQ][innerColQ + t * numThreadsBlocktile] = Q[batch_offset + id];
      }
      else {
        Q_i[innerRowQ][innerColQ + t * numThreadsBlocktile] = 0.0;
      }
    }

    __syncthreads();

    // scratchpad register for register-tiling (coarsening of the matrix mults)
    float regM[TM] = {0.0};
    float regN[TN] = {0.0};

    for (int j = 0; j < T_c; j++) { // iterate of ver the chunks of K and V
        float threadResults[TM * TN] = {0.0}; // storing the intermediate outputs
        
        for (int t=0; t<num_tiles; t++){
            // load K_j and V_j, thread idx, idy loads idy,idx
            // we load a tile
            for (int i=0; i<B_r; i+=strideK){
                // load Q, K and V in tiles (for now we are loading the full V)
                if (not CACHE_Q){Q_i[innerRowK+i][innerColK] = Q[batch_offset + (innerRowK + blockIdx.y * B_r) * d  + i * d + innerColK + t * B_c];
                } // if you cache Q over whole d then remove this line
                id = (innerRowK + j * B_c) * d + i * d + innerColK + t * B_c;
                if (id < d*seq_len){
                    K_j[innerRowK+i][innerColK] = K[batch_offset + id];
                    //V_j[innerRowK+i][innerColK+t*B_c] = V[batch_offset + id];
                } else {
                    K_j[innerRowK+i][innerColK] = 0.0;
                    //V_j[innerRowK+i][innerColK+t*B_c] = 0.0;
                }
        
            }
            __syncthreads();
        
            for (int dd=0; dd<BK; dd++){ // load elements of Q_i and K_j^T into registers
                for (uint i = 0; i < TM; ++i) {
                    if (CACHE_Q){
                        regM[i] = Q_i[(threadRow * TM + i)][dd+t*BK]; // uncomment if you cache Q over full d
                    } else {
                        regM[i] = Q_i[(threadRow * TM + i)][dd];
                    }
                }
                for (uint i = 0; i < TN; ++i) {
                    regN[i] = K_j[threadCol * TN + i][dd];
                }
                for (uint resIdxM = 0; resIdxM < TM; ++resIdxM) {
                    for (uint resIdxN = 0; resIdxN < TN; ++resIdxN) {
                        threadResults[resIdxM * TN + resIdxN] += regM[resIdxM] * regN[resIdxN];
                    }
                }
            }
            __syncthreads();
        }
        

        // store the results in S_i, account for causal masking
        for (uint resIdxM = 0; resIdxM < TM; ++resIdxM) {
            for (uint resIdxN = 0; resIdxN < TN; ++resIdxN) {
                    S_i[(threadRow * TM + resIdxM)][threadCol * TN + resIdxN] = threadResults[resIdxM * TN + resIdxN] *scaling;
            }
        }
        __syncthreads();

        for (int i=0;i<TM;++i){
            last_m[i] = m_i[i];
            float m = m_i[i];
            for (int jj = 0; jj < B_c; jj += 1) {
                if (m < S_i[threadRow*TM+i][jj]) {
                    m = S_i[threadRow*TM+i][jj];
                }
            }
            m_i[i] = m;
        }

        // 2) renormalize current O
        if (j > 0) {
            for (int t = 0; t < num_tiles; t++){
                for (int i=0;i<TM;++i){
                    for (int jj=0;jj<TN;++jj){
                        O_i[t*TN*TM + i*TN + jj] *= exp(last_m[i] - m_i[i]);
                    }
                }
            }
        }

        // 3) renormalize the sum l_i
        for (int i=0;i<TM;++i){
            l_i[i] *= exp(last_m[i] - m_i[i]);
        }

        for (int t = 0; t < num_tiles; t++){
            // load V
            __syncthreads();
            for (int i=0; i<B_r; i+=strideK){
                id = (innerRowK + j * B_c) * d + i * d + innerColK + t * B_c;
                if (id < d*seq_len){
                    V_j[innerRowK+i][innerColK] = V[batch_offset + id];
                } else {
                    V_j[innerRowK+i][innerColK] = 0.0;
                }
            }
            __syncthreads();

            for (int dd = 0; dd < B_c; dd++) {
                for (int ii = 0; ii < TN; ii++){
                    regM[ii] = exp(S_i[threadRow*TM+ii][dd] - m_i[ii]);
                    if (t==0){
                        l_i[ii] += regM[ii];
                    }
                    regN[ii] = V_j[dd][threadCol * TN + ii];
                }
                for (int ii=0;ii<TN;ii++){
                    for (int jj=0;jj<TM;jj++){ // calculate output elements
                        regN[jj] = V_j[dd][threadCol * TN + jj];
                        O_i[t*TN*TM + ii*TM + jj] += regM[ii] * regN[jj];
                    }
                }
            }
            __syncthreads();
        }
    }

    // normalize by the output sum and write to out matrix
    for (int t = 0; t < num_tiles; t++){
        for (int ii=0;ii<TM;ii++){
            for (int jj=0;jj<TN;jj++){
                if(blockIdx.y*B_r+threadRow*TM+ii < seq_len){
                    out[batch_offset + (blockIdx.y * B_r + threadRow*TM + ii) * d + t * B_c + threadCol*TN + jj] = O_i[t*TN*TM+ii*TM+jj] / l_i[ii];
                }
            }
        } 
    }
}

void run_flash_tiled_coarse(torch::Tensor O,
                            torch::Tensor O_l,
                            torch::Tensor K_d,
                            torch::Tensor Q_d,
                            torch::Tensor V_d,
                            int batch_size,   // "virtual" batch = B*H
                            int seq_len)
{
    dim3 blockDim(B_r / TN, B_c / TM);
    dim3 gridDim(batch_size, (seq_len + B_r - 1) / B_r);

    flash_tiled_coarse<<<gridDim, blockDim>>>(
        O.data_ptr<float>(),
        O_l.data_ptr<float>(),
        K_d.data_ptr<float>(),
        Q_d.data_ptr<float>(),
        V_d.data_ptr<float>(),
        1.0f / sqrtf((float)d),
        seq_len * d,                       // batch-stride (S·D)
        (seq_len + B_r - 1) / B_r,         // T_r
        (seq_len + B_c - 1) / B_c,         // T_c
        seq_len);

    hipDeviceSynchronize();
}

torch::Tensor forward(torch::Tensor Q_d,
                      torch::Tensor K_d,
                      torch::Tensor V_d)
{
    TORCH_CHECK(Q_d.is_cuda() && K_d.is_cuda() && V_d.is_cuda(),
                "All tensors must be CUDA");

    /* -------- shapes --------
       Expected: (B, H, S, D) with D == d (compile-time constant 64)
    -------------------------------- */
    TORCH_CHECK(Q_d.dim() == 4, "Q must be (B,H,S,D)");
    int B = Q_d.size(0);
    int H = Q_d.size(1);
    int S = Q_d.size(2);
    TORCH_CHECK(Q_d.size(3) == d,
                "Last dimension must be ", d);

    /* Merge batch and head into a single "virtual batch" */
    int virtual_batch = B * H;
    auto Qv = Q_d.contiguous().view({virtual_batch, S, d});
    auto Kv = K_d.contiguous().view({virtual_batch, S, d});
    auto Vv = V_d.contiguous().view({virtual_batch, S, d});

    /* Allocate outputs with the same merged layout */
    auto O  = torch::zeros({virtual_batch, S, d},
                           Q_d.options());
    auto O_l = torch::zeros({virtual_batch, S},
                            Q_d.options());

    run_flash_tiled_coarse(O, O_l, Kv, Qv, Vv,
                           virtual_batch, S);

    /* Reshape back to (B, H, S, D) and return */
    return O.view({B, H, S, d});
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &forward,
          "Flash-tiled coarse attention (supports B,H,S,D)");
}